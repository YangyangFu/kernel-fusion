#include "hip/hip_runtime.h"
#include "kernel_fusion/kernels/kernels.hpp"
#include <torch/torch.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <cmath>

class FusionValidation {
public:
    FusionValidation() {
        torch_device_ = torch::Device(torch::kCUDA, 0);
        hipStreamCreate(&stream_);
    }
    
    ~FusionValidation() {
        hipStreamDestroy(stream_);
    }
    
    bool validate_all() {
        std::cout << "=== Validating Kernel Fusion Correctness ===" << std::endl;
        
        bool all_passed = true;
        all_passed &= validate_add_relu();
        all_passed &= validate_add_gelu();
        all_passed &= validate_mul_silu();
        all_passed &= validate_bias_activation();
        
        if (all_passed) {
            std::cout << "\n✅ All fusion validations PASSED!" << std::endl;
            std::cout << "Kernel fusion produces identical results to separate operations." << std::endl;
        } else {
            std::cout << "\n❌ Some fusion validations FAILED!" << std::endl;
        }
        
        return all_passed;
    }

private:
    torch::Device torch_device_;
    hipStream_t stream_;
    
    bool validate_add_relu() {
        std::cout << "\nValidating ADD + RELU fusion..." << std::endl;
        
        const size_t n = 10000;
        
        // Create random test data
        std::vector<float> h_a(n), h_b(n);
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dis(-3.0f, 3.0f);
        
        for (size_t i = 0; i < n; ++i) {
            h_a[i] = dis(gen);
            h_b[i] = dis(gen);
        }
        
        // 1. Our fused kernel result
        std::vector<float> fused_result = compute_fused_add_relu(h_a, h_b);
        
        // 2. PyTorch separate operations result
        std::vector<float> pytorch_result = compute_pytorch_add_relu(h_a, h_b);
        
        // 3. CPU reference result
        std::vector<float> cpu_result(n);
        for (size_t i = 0; i < n; ++i) {
            cpu_result[i] = std::max(0.0f, h_a[i] + h_b[i]);
        }
        
        // Compare results
        bool fused_vs_cpu = compare_arrays(fused_result, cpu_result, "Fused vs CPU");
        bool fused_vs_pytorch = compare_arrays(fused_result, pytorch_result, "Fused vs PyTorch");
        
        return fused_vs_cpu && fused_vs_pytorch;
    }
    
    bool validate_add_gelu() {
        std::cout << "\nValidating ADD + GELU fusion..." << std::endl;
        
        const size_t n = 10000;
        
        std::vector<float> h_a(n), h_b(n);
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dis(-2.0f, 2.0f);
        
        for (size_t i = 0; i < n; ++i) {
            h_a[i] = dis(gen);
            h_b[i] = dis(gen);
        }
        
        // Our fused kernel
        std::vector<float> fused_result = compute_fused_add_gelu(h_a, h_b);
        
        // PyTorch reference
        std::vector<float> pytorch_result = compute_pytorch_add_gelu(h_a, h_b);
        
        // CPU reference (GELU approximation)
        std::vector<float> cpu_result(n);
        for (size_t i = 0; i < n; ++i) {
            float x = h_a[i] + h_b[i];
            float x3 = x * x * x;
            float inner = 0.7978845608f * (x + 0.044715f * x3);
            cpu_result[i] = 0.5f * x * (1.0f + std::tanh(inner));
        }
        
        bool fused_vs_cpu = compare_arrays(fused_result, cpu_result, "Fused vs CPU", 1e-5f);
        bool fused_vs_pytorch = compare_arrays(fused_result, pytorch_result, "Fused vs PyTorch", 1e-5f);
        
        return fused_vs_cpu && fused_vs_pytorch;
    }
    
    bool validate_mul_silu() {
        std::cout << "\nValidating MUL + SiLU fusion..." << std::endl;
        
        const size_t n = 10000;
        
        std::vector<float> h_a(n), h_b(n);
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dis(-2.0f, 2.0f);
        
        for (size_t i = 0; i < n; ++i) {
            h_a[i] = dis(gen);
            h_b[i] = dis(gen);
        }
        
        // Our fused kernel
        std::vector<float> fused_result = compute_fused_mul_silu(h_a, h_b);
        
        // PyTorch reference
        std::vector<float> pytorch_result = compute_pytorch_mul_silu(h_a, h_b);
        
        // CPU reference
        std::vector<float> cpu_result(n);
        for (size_t i = 0; i < n; ++i) {
            float x = h_a[i] * h_b[i];
            cpu_result[i] = x / (1.0f + std::exp(-x));
        }
        
        bool fused_vs_cpu = compare_arrays(fused_result, cpu_result, "Fused vs CPU", 1e-6f);
        bool fused_vs_pytorch = compare_arrays(fused_result, pytorch_result, "Fused vs PyTorch", 1e-6f);
        
        return fused_vs_cpu && fused_vs_pytorch;
    }
    
    bool validate_bias_activation() {
        std::cout << "\nValidating BIAS + RELU fusion..." << std::endl;
        
        const size_t n = 10000;
        const size_t bias_size = 128;
        
        std::vector<float> h_input(n), h_bias(bias_size);
        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dis(-1.0f, 1.0f);
        
        for (size_t i = 0; i < n; ++i) {
            h_input[i] = dis(gen);
        }
        for (size_t i = 0; i < bias_size; ++i) {
            h_bias[i] = dis(gen);
        }
        
        // Our fused kernel
        std::vector<float> fused_result = compute_fused_bias_relu(h_input, h_bias);
        
        // CPU reference
        std::vector<float> cpu_result(n);
        for (size_t i = 0; i < n; ++i) {
            size_t bias_idx = i % bias_size;
            float biased = h_input[i] + h_bias[bias_idx];
            cpu_result[i] = std::max(0.0f, biased);
        }
        
        bool validation_passed = compare_arrays(fused_result, cpu_result, "Fused vs CPU");
        
        return validation_passed;
    }
    
    std::vector<float> compute_fused_add_relu(const std::vector<float>& a, const std::vector<float>& b) {
        size_t n = a.size();
        
        float *d_a, *d_b, *d_output;
        hipMalloc(&d_a, n * sizeof(float));
        hipMalloc(&d_b, n * sizeof(float));
        hipMalloc(&d_output, n * sizeof(float));
        
        hipMemcpy(d_a, a.data(), n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b.data(), n * sizeof(float), hipMemcpyHostToDevice);
        
        dim3 block_size(256);
        dim3 grid_size((n + block_size.x - 1) / block_size.x);
        
        kf::kernels::elementwise::add_activation_kernel<float><<<grid_size, block_size, 0, stream_>>>(
            d_a, d_b, d_output, n, KF_ACTIVATION_RELU);
        
        hipDeviceSynchronize();
        
        std::vector<float> result(n);
        hipMemcpy(result.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost);
        
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_output);
        
        return result;
    }
    
    std::vector<float> compute_fused_add_gelu(const std::vector<float>& a, const std::vector<float>& b) {
        size_t n = a.size();
        
        float *d_a, *d_b, *d_output;
        hipMalloc(&d_a, n * sizeof(float));
        hipMalloc(&d_b, n * sizeof(float));
        hipMalloc(&d_output, n * sizeof(float));
        
        hipMemcpy(d_a, a.data(), n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b.data(), n * sizeof(float), hipMemcpyHostToDevice);
        
        dim3 block_size(256);
        dim3 grid_size((n + block_size.x - 1) / block_size.x);
        
        kf::kernels::elementwise::add_activation_kernel<float><<<grid_size, block_size, 0, stream_>>>(
            d_a, d_b, d_output, n, KF_ACTIVATION_GELU);
        
        hipDeviceSynchronize();
        
        std::vector<float> result(n);
        hipMemcpy(result.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost);
        
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_output);
        
        return result;
    }
    
    std::vector<float> compute_fused_mul_silu(const std::vector<float>& a, const std::vector<float>& b) {
        size_t n = a.size();
        
        float *d_a, *d_b, *d_output;
        hipMalloc(&d_a, n * sizeof(float));
        hipMalloc(&d_b, n * sizeof(float));
        hipMalloc(&d_output, n * sizeof(float));
        
        hipMemcpy(d_a, a.data(), n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_b, b.data(), n * sizeof(float), hipMemcpyHostToDevice);
        
        dim3 block_size(256);
        dim3 grid_size((n + block_size.x - 1) / block_size.x);
        
        kf::kernels::elementwise::mul_activation_kernel<float><<<grid_size, block_size, 0, stream_>>>(
            d_a, d_b, d_output, n, KF_ACTIVATION_SILU);
        
        hipDeviceSynchronize();
        
        std::vector<float> result(n);
        hipMemcpy(result.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost);
        
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_output);
        
        return result;
    }
    
    std::vector<float> compute_fused_bias_relu(const std::vector<float>& input, const std::vector<float>& bias) {
        size_t n = input.size();
        size_t bias_size = bias.size();
        
        float *d_input, *d_bias, *d_output;
        hipMalloc(&d_input, n * sizeof(float));
        hipMalloc(&d_bias, bias_size * sizeof(float));
        hipMalloc(&d_output, n * sizeof(float));
        
        hipMemcpy(d_input, input.data(), n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_bias, bias.data(), bias_size * sizeof(float), hipMemcpyHostToDevice);
        
        dim3 block_size(256);
        dim3 grid_size((n + block_size.x - 1) / block_size.x);
        
        kf::kernels::elementwise::bias_activation_kernel<float><<<grid_size, block_size, 0, stream_>>>(
            d_input, d_bias, d_output, n, bias_size, KF_ACTIVATION_RELU);
        
        hipDeviceSynchronize();
        
        std::vector<float> result(n);
        hipMemcpy(result.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost);
        
        hipFree(d_input);
        hipFree(d_bias);
        hipFree(d_output);
        
        return result;
    }
    
    std::vector<float> compute_pytorch_add_relu(const std::vector<float>& a, const std::vector<float>& b) {
        size_t n = a.size();
        
        auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch_device_);
        auto tensor_a = torch::from_blob((void*)a.data(), {static_cast<long>(n)}, torch::kFloat32).to(torch_device_);
        auto tensor_b = torch::from_blob((void*)b.data(), {static_cast<long>(n)}, torch::kFloat32).to(torch_device_);
        
        auto sum = torch::add(tensor_a, tensor_b);
        auto result_tensor = torch::relu(sum);
        
        auto cpu_tensor = result_tensor.cpu();
        std::vector<float> result(n);
        std::memcpy(result.data(), cpu_tensor.data_ptr(), n * sizeof(float));
        
        return result;
    }
    
    std::vector<float> compute_pytorch_add_gelu(const std::vector<float>& a, const std::vector<float>& b) {
        size_t n = a.size();
        
        auto tensor_a = torch::from_blob((void*)a.data(), {static_cast<long>(n)}, torch::kFloat32).to(torch_device_);
        auto tensor_b = torch::from_blob((void*)b.data(), {static_cast<long>(n)}, torch::kFloat32).to(torch_device_);
        
        auto sum = torch::add(tensor_a, tensor_b);
        auto result_tensor = torch::gelu(sum);
        
        auto cpu_tensor = result_tensor.cpu();
        std::vector<float> result(n);
        std::memcpy(result.data(), cpu_tensor.data_ptr(), n * sizeof(float));
        
        return result;
    }
    
    std::vector<float> compute_pytorch_mul_silu(const std::vector<float>& a, const std::vector<float>& b) {
        size_t n = a.size();
        
        auto tensor_a = torch::from_blob((void*)a.data(), {static_cast<long>(n)}, torch::kFloat32).to(torch_device_);
        auto tensor_b = torch::from_blob((void*)b.data(), {static_cast<long>(n)}, torch::kFloat32).to(torch_device_);
        
        auto product = torch::mul(tensor_a, tensor_b);
        auto result_tensor = torch::silu(product);
        
        auto cpu_tensor = result_tensor.cpu();
        std::vector<float> result(n);
        std::memcpy(result.data(), cpu_tensor.data_ptr(), n * sizeof(float));
        
        return result;
    }
    
    bool compare_arrays(const std::vector<float>& a, const std::vector<float>& b, 
                       const std::string& desc, float tolerance = 1e-6f) {
        if (a.size() != b.size()) {
            std::cout << "  ❌ " << desc << ": Size mismatch" << std::endl;
            return false;
        }
        
        float max_error = 0.0f;
        size_t error_count = 0;
        size_t n = a.size();
        
        for (size_t i = 0; i < n; ++i) {
            float diff = std::abs(a[i] - b[i]);
            float rel_tolerance = tolerance * std::max(std::abs(a[i]), std::abs(b[i]));
            float effective_tolerance = std::max(rel_tolerance, tolerance);
            
            if (diff > effective_tolerance) {
                error_count++;
                max_error = std::max(max_error, diff);
                
                if (error_count <= 3) {
                    std::cout << "    Error at " << i << ": " << a[i] << " vs " << b[i] 
                              << " (diff: " << diff << ")" << std::endl;
                }
            }
        }
        
        if (error_count > 0) {
            std::cout << "  ❌ " << desc << ": " << error_count << "/" << n 
                      << " mismatches, max error: " << max_error << std::endl;
            return false;
        } else {
            std::cout << "  ✅ " << desc << ": All values match within tolerance" << std::endl;
            return true;
        }
    }
};

int main() {
    try {
        FusionValidation validator;
        bool success = validator.validate_all();
        return success ? 0 : 1;
    } catch (const std::exception& e) {
        std::cerr << "Validation failed: " << e.what() << std::endl;
        return 1;
    }
}
